﻿#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define DIMU 4
#define DIMV 4
#define DIMW 4
#define TYPE double

__global__ void gpuVectorAdd(TYPE* bufferIn1, TYPE* bufferIn2, TYPE* bufferOut, int bufferSize)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < bufferSize) bufferOut[tid] = bufferIn1[tid] + bufferIn2[tid];
}

int main()
{
	TYPE* cpuMatrixA;
	TYPE* cpuMatrixB;
	TYPE* cpuMatrixC;
	TYPE* gpuMatrixA;
	TYPE* gpuMatrixB;
	TYPE* gpuMatrixC;

	srand(time(NULL));

	cpuMatrixA = (TYPE*)malloc(DIMU * DIMV * sizeof(TYPE));
	cpuMatrixB = (TYPE*)malloc(DIMV * DIMW * sizeof(TYPE));
	cpuMatrixC = (TYPE*)malloc(DIMU * DIMW * sizeof(TYPE));

	hipMalloc((void**)&gpuMatrixA, DIMU * DIMV * sizeof(TYPE));
	hipMalloc((void**)&gpuMatrixB, DIMV * DIMW * sizeof(TYPE));
	hipMalloc((void**)&gpuMatrixC, DIMU * DIMW * sizeof(TYPE));

	for (int i = 0; i != DIMU * DIMV; ++i) cpuMatrixA[i] = (TYPE)rand() / (TYPE)RAND_MAX;
	for (int i = 0; i != DIMV * DIMW; ++i) cpuMatrixB[i] = (TYPE)rand() / (TYPE)RAND_MAX;

	hipMemcpy(gpuMatrixA, cpuMatrixA, DIMU * DIMV * sizeof(TYPE), hipMemcpyHostToDevice);
	hipMemcpy(gpuMatrixB, cpuMatrixB, DIMV * DIMW * sizeof(TYPE), hipMemcpyHostToDevice);

	//int threads = 64;
	//int blocks  = (SIZE + threads - 1) / threads; // TODO

	//gpuVectorAdd<<<blocks, threads>>>(gpuBufferIn1, gpuBufferIn2, gpuBufferOut, SIZE); // TODO
	hipDeviceSynchronize();

	hipMemcpy(cpuMatrixC, gpuMatrixC, DIMU * DIMW * sizeof(TYPE), hipMemcpyDeviceToHost);

	int errorCount = 0;

	for (int i = 0; i != DIMU * DIMW; ++i)
	{
		int u = i / DIMW;
		int w = i % DIMW;

		TYPE value = (TYPE)0;

		for (int v = 0; v != DIMV; ++v)
		{
			value += cpuMatrixA[v + u * DIMV] * cpuMatrixB[w + u * DIMW];
		}

		errorCount += (value != cpuMatrixC[i]);
	}

	printf("Matrix A: (%d, %d)\n", DIMU, DIMV);
	printf("Matrix B: (%d, %d)\n", DIMV, DIMW);
	printf("Matrix C: (%d, %d)\n", DIMU, DIMW);
	printf("Errors:   %d\n", errorCount);

	hipFree(gpuMatrixA);
	hipFree(gpuMatrixB);
	hipFree(gpuMatrixC);

	free(cpuMatrixA);
	free(cpuMatrixB);
	free(cpuMatrixC);

	return 0;
}